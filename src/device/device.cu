#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#include "device.h"
#include "launcher.h"

void launcher_test() {
    auto l = Launcher::GetInstance();
    int count = l->device_count();
    std::cout << "device count: " << count << std::endl;
}
